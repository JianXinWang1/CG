#include "hip/hip_runtime.h"
﻿#include "helper.h"
#include "hip/hip_vector_types.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>

inline __host__ __device__ float* barycentric_gpu(float* A, float* B, float* C, float* p)
{
    float* res = new float[3];
    //别丢了分母等于0的情况
    if ((-(A[0] - B[0]) * (C[1] - B[1]) + (A[1] - B[1]) * (C[0] - B[0])) == 0) {
        res[0] = 1.0;
        res[1] = 0.0;
        res[2] = 0.0;
        return res;
    }
    if (-(B[0] - C[0]) * (A[1] - C[1]) + (B[1] - C[1]) * (A[0] - C[0]) == 0) {
        res[0] = 1.0;
        res[1] = 0.0;
        res[2] = 0.0;
        return res;
    }
    float alpha = (-(p[0] - B[0]) * (C[1] - B[1]) + (p[1] - B[1]) * (C[0] - B[0])) / (-(A[0] - B[0]) * (C[1] - B[1]) + (A[1] - B[1]) * (C[0] - B[0]));
    float beta = (-(p[0] - C[0]) * (A[1] - C[1]) + (p[1] - C[1]) * (A[0] - C[0])) / (-(B[0] - C[0]) * (A[1] - C[1]) + (B[1] - C[1]) * (A[0] - C[0]));
    float gamma = 1 - alpha - beta;

    res[0] = alpha;
    res[1] = beta;
    res[2] = gamma;
    return res;
}

__global__ void gpushaderkernel(VertexData* fp_vertex_device, ComputeData* fp_compute_device, GputoCpuData* pgputocpu, int n, int width, int height) {
	const int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) {
        return;
    }
    
    float (*ps)[3] = fp_vertex_device[id].mvp_ps;
    float* ws = fp_vertex_device[id].ws;
    float(*ps_prior)[3] = fp_vertex_device[id].model_ps;
    float(*ns)[3] = fp_vertex_device[id].ns;
    float(*uvs)[2] = fp_vertex_device[id].uvs;

    float* depthbuffer = fp_compute_device->depthbuffer;
    float* camera_point = fp_compute_device->camera_point;
    float* light_dir = fp_compute_device->light_dir;

    float* colorbuffer_x = pgputocpu->colorbuffer_x;
    float* colorbuffer_y = pgputocpu->colorbuffer_y;
    float* colorbuffer_indensity = pgputocpu->colorbuffer_indensity;
    float* colorbuffer_blinphong = pgputocpu->colorbuffer_blinphong;
    bool* idxs = pgputocpu->idxs;


    // 到图像坐标
    float x1 = (ps[0][0] + 1.) * width / 2., y1 = (ps[0][1] + 1.) * height / 2.;
    float x2 = (ps[1][0] + 1.) * width / 2., y2 = (ps[1][1] + 1.) * height / 2.;
    float x3 = (ps[2][0] + 1.) * width / 2., y3 = (ps[2][1] + 1.) * height / 2.;

    //printf("%f %f %f %f %f \n", ps[0][0], ws[0], ps_prior[0][0], ns[0][0], uvs[2][0]);

    // 图像坐标系包围框
    float x_min = min(x1, x2);
    x_min = min(x_min, x3);

    float x_max = max(x1, x2);
    x_max = max(x_max, x3);

    float y_min = min(y1, y2);
    y_min = min(y_min, y3);

    float y_max = max(y1, y2);
    y_max = max(y_max, y3);

    for (int x = x_min - 1; x <= x_max + 1; x++) {
        for (int y = y_min - 1; y <= y_max + 1; y++) {
            if (x < 0 || x > width || y < 0 || y > height) {
                continue;
            }
            float A[2] = { x1, y1 };
            float B[2] = { x2, y2 };
            float C[2] = { x3, y3 };
            float p[2] = { x + 0.5, y + 0.5 };
            float AB = (B[0] - A[0]) * (p[1] - A[1]) - (B[1] - A[1]) * (p[0] - A[0]);
            float BC = (C[0] - B[0]) * (p[1] - B[1]) - (C[1] - B[1]) * (p[0] - B[0]);
            float CA = (A[0] - C[0]) * (p[1] - C[1]) - (A[1] - C[1]) * (p[0] - C[0]);
            if ((AB > 0 && BC > 0 && CA > 0) || (AB < 0 && BC < 0 && CA < 0)) {
                float* bary_centric = barycentric_gpu(A, B, C, p);
                // 透视矫正
                float k = 1.0 / (bary_centric[0] * ws[0] + bary_centric[1] * ws[1] + bary_centric[2] * ws[2]);
                float alpha = bary_centric[0] / (ws[0] * k);
                float beta = bary_centric[1] / (ws[1] * k);
                float gama = bary_centric[2] / (ws[2] * k);

                float p_depth = (alpha * ps[0][2] + beta * ps[1][2] + gama * ps[2][2]) / 2.0 + 0.5;
                int idx = y * height + x;
                if (depthbuffer[idx] < p_depth) {
                    depthbuffer[idx] = p_depth;

                    float3 ps_prior1_cu = make_float3(ps_prior[0][0], ps_prior[0][1], ps_prior[0][2]);
                    float3 ps_prior2_cu = make_float3(ps_prior[1][0], ps_prior[1][1], ps_prior[1][2]);
                    float3 ps_prior3_cu = make_float3(ps_prior[2][0], ps_prior[2][1], ps_prior[2][2]);
                    float3 interpertion_point = ps_prior1_cu * alpha + ps_prior2_cu * beta + ps_prior3_cu * gama;

                    float3 camera_point_cu = make_float3(camera_point[0], camera_point[1], camera_point[2]);
                    float3 eye_light = camera_point_cu - interpertion_point;
                    eye_light = normalize(eye_light);

                    float3 n1_cu = normalize(make_float3(ns[0][0], ns[0][1], ns[0][2]));
                    float3 n2_cu = normalize(make_float3(ns[1][0], ns[1][1], ns[1][2]));
                    float3 n3_cu = normalize(make_float3(ns[2][0], ns[2][1], ns[2][2]));

                    float3 normal = n1_cu * alpha + n2_cu * beta + n3_cu * gama;
                    normal = normalize(normal);

                    float3 light_dir_cu = normalize(make_float3(light_dir[0], light_dir[1], light_dir[2]));
                    float3 half_v = eye_light + light_dir_cu;
                    half_v = normalize(half_v);

                    float blin_phong = pow(max(0.f, dot(half_v, normal)), 16);
                    float indensity = max(0.f, dot(normal, light_dir_cu));

                    float2 color_index_cu = make_float2(alpha * uvs[0][0] + beta * uvs[1][0] + gama * uvs[2][0], alpha * uvs[0][1] + beta * uvs[1][1] + gama * uvs[2][1]);

                    colorbuffer_x[idx] = color_index_cu.x;
                    colorbuffer_y[idx] = color_index_cu.y;
                    colorbuffer_indensity[idx] = indensity;
                    colorbuffer_blinphong[idx] = blin_phong;
                    idxs[idx] = true;
                    //printf("%d %f %f %f %f\n", idx, color_index_cu.x, color_index_cu.y, indensity, blin_phong);
                }
            }
        }
    }

}

GputoCpuData* gpu_memcpy(VertexData* pgpu_vertex_data, ComputeData* pgpu_computedata, GputoCpuData* pgputocpu, int n, int width, int height) {
    hipSetDevice(0);

    VertexData* fp_vertex_device;
    hipMalloc((void**)&fp_vertex_device, n * sizeof(VertexData));

    ComputeData* fp_compute_device;
    hipMalloc((void**)&fp_compute_device, sizeof(ComputeData));

    GputoCpuData* fp_gputocpu;
    hipMalloc((void**)&fp_gputocpu, sizeof(GputoCpuData));

    hipMemcpy(fp_vertex_device, pgpu_vertex_data, n * sizeof(VertexData), hipMemcpyHostToDevice);
    hipMemcpy(fp_compute_device, pgpu_computedata, sizeof(ComputeData), hipMemcpyHostToDevice);
    auto stat = hipMemcpy(fp_gputocpu, pgputocpu, sizeof(GputoCpuData), hipMemcpyHostToDevice);

    int threads = 256, blocks = (n + threads - 1) / threads;

    gpushaderkernel << <blocks, threads >> > (fp_vertex_device, fp_compute_device, fp_gputocpu, n, width, height);
    hipDeviceSynchronize();

    GputoCpuData res_gputocpu;
    stat = hipMemcpy(&res_gputocpu,  fp_gputocpu, sizeof(GputoCpuData), hipMemcpyDeviceToHost);

    return &res_gputocpu;
}
